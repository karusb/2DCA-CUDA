#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>  
#include <stdio.h>
#include <algorithm>
#define PanelW 1024
#define PanelH 728
//

//#include <GL\GL.h>
//#include <GL\GLU.h>
//#include <GL\glut.h>
//#include <cudaGL.h>
#include <GL\glew.h>
//#include <GL\glxew.h>
//#include <GL\wglew.h>
#include <GL\freeglut.h>
#include "cuda_gl_interop.h"
#include "Header.h"
#define nCPUGRAPHICS



hipGraphicsResource* cudaPboResource = nullptr;
GLuint GLtexture;
GLuint GLbufferID;
uchar4 *d_texturedata = nullptr;
uchar4 *d_bufferdata = nullptr;
uchar4 *GLout = nullptr;
bool *d_CAGrid = nullptr;

hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size, unsigned int WorldH, unsigned int WorldW,unsigned int gen,int*argc,char**argv);
/*
__device__ int NeighboursEval(bool *CAGrid, int x, int y,int WorldH,int WorldW)
{

	const unsigned int pos = (y*WorldW) + x;
	const unsigned int colup = x + ( (y - 1)*WorldW);
	const unsigned int coldwn = x + ((y + 1)*WorldW);
	if (x >= 0 && y >= 0 && x <= WorldW && y <= WorldH)
	{
		return  CAGrid[pos + 1] +
		 CAGrid[pos - 1]+
		 CAGrid[colup - 1]+
		 CAGrid[colup]+
		 CAGrid[colup + 1]+
		 CAGrid[coldwn - 1]+
		 CAGrid[coldwn]+
		 CAGrid[coldwn + 1];
	}
	return 0;
}
__device__ int NeighboursEval_Global(bool *CAGrid, int GlobalID, int WorldH, int WorldW)
{

	unsigned int colup = GlobalID - ((blockIdx.y - 1)*blockDim.x);
	unsigned int coldwn = GlobalID + ((blockIdx.y + 1)*blockDim.x);
	if (GlobalID > 0 && (GlobalID < WorldH * WorldW) && (GlobalID < blockDim.y * blockDim.x))
	{
		return  CAGrid[GlobalID + 1] +
			CAGrid[GlobalID - 1] +
			CAGrid[colup - 1] +
			CAGrid[colup] +
			CAGrid[colup + 1] +
			CAGrid[coldwn - 1] +
			CAGrid[coldwn] +
			CAGrid[coldwn + 1];
	}
	return 0;
}
__device__ int getGlobalIdx()
{
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}
__global__ void NextDumbKernel(bool *CAGrid, bool *NextCAGrid)
{
//int id = getGlobalIdx();
//int neighbours = 0;

}
*/

void OpenGLHelper(unsigned int width,unsigned int height)
{


	//float4 *h_texturedata = new float4[width * height];
	//float4 *h_bufferdata = new float4[width * height];

	glGenTextures(1, &GLtexture);
	glBindTexture(GL_TEXTURE_2D, GLtexture);
	
	// set basic parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// Create texture data (4-component unsigned byte)
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, d_texturedata);

	// Unbind the texture
	glBindTexture(GL_TEXTURE_2D, 0);

	
	glGenBuffers(1, &GLbufferID);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, GLbufferID);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * sizeof(uchar4), d_bufferdata, GL_STREAM_COPY);

	//glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	
	hipError_t result = hipGraphicsGLRegisterBuffer(&cudaPboResource, GLbufferID,
		cudaGraphicsMapFlagsWriteDiscard);


}
bool initGLUT(int* argc, char** argv,unsigned int width,unsigned int height) {
	glutInit(argc, argv);  // Create GL context.
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(width, height);
	glutCreateWindow("Interesting Life Story");

	glewInit();

	if (!glewIsSupported("GL_VERSION_2_0")) {
		printf( "ERROR: Support for necessary OpenGL extensions missing.\n");
		return false;
	}

	glutReportErrors();
	return true;
}
void drawTexture(unsigned int width,unsigned int height) {
	glColor3f(1.0f, 1.0f, 1.0f);
	glBindTexture(GL_TEXTURE_2D, GLtexture);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, GLbufferID);

	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glEnable(GL_TEXTURE_2D);
	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(0.0f, 0.0f);
	glTexCoord2f(1.0f, 0.0f);
	glVertex2f(float(width), 0.0f);
	glTexCoord2f(1.0f, 1.0f);
	glVertex2f(float(width), float(height));
	glTexCoord2f(0.0f, 1.0f);
	glVertex2f(0.0f, float(height));
	glEnd();
	glDisable(GL_TEXTURE_2D);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glBindTexture(GL_TEXTURE_2D, 0);
}

__global__ void NextGenKernel(bool *CAGrid, bool *NextCAGrid,int WorldH,int WorldW)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y*blockDim.y + threadIdx.y) * WorldH;
	int neighbours = 0;

	unsigned int colup = id - ((blockIdx.y - 1)*blockDim.x);
	unsigned int coldwn = id + ((blockIdx.y + 1)*blockDim.x);
	if (id < (WorldH - 2) * (WorldW - 2) && id>=1)
	{
		// TODO WHY CODE 77 HERE - SOLVED ACCESS
		//neighbours = NeighboursEval_Global(CAGrid, id, WorldH, WorldW);
		// NEIGHBOUR ADDITION

			neighbours =  CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];


		// END OF NEIGHBOUR ADDITION



		if (neighbours < 2)
		{
			NextCAGrid[id] = 0;
		}
		else if (neighbours > 3)
		{
			NextCAGrid[id] = 0;
		}
		else if (neighbours == 3)
		{
			NextCAGrid[id] = 1;
		}
	}
}
__global__ void GLKernel(uchar4 *d_buf,bool *CAGrid, int WorldH, int WorldW)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y*blockDim.y + threadIdx.y) * WorldH;
	if (id < (WorldH - 2) * (WorldW - 2) && id >= 1)
	{
		if (CAGrid[id])
		{
			d_buf[id].w = 255;
			d_buf[id].x = 191;
			d_buf[id].y = 173;
			d_buf[id].z = 134;
		}
		else
		{
			d_buf[id].w = 255;
			d_buf[id].x = 0;
			d_buf[id].y = 0;
			d_buf[id].z = 0;
		}
	}
}
void displayfunc()
{
	int WorldW = PanelW;
	int WorldH = PanelH;
	dim3 kernelwsize(WorldW, WorldH);
	dim3 kernelbsize(1);
	//LAUNCH OTHER KERNEL HERE
	hipGraphicsMapResources(1, &cudaPboResource, 0);
	size_t num_bytes;
	hipError_t cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&GLout,
		&num_bytes, cudaPboResource);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Resource Mapping Error: %s\n", hipGetErrorString(cudaStatus));
		}
	//cudaGLSetGLDevice(0);
	GLKernel << < kernelwsize, kernelbsize >> > (GLout, d_CAGrid, WorldH, WorldW);
	hipGraphicsUnmapResources(1, &cudaPboResource, 0);
	drawTexture(WorldW, WorldH);
	glutSwapBuffers();
	
}
int main(int argc,char** argv)
{
    

	const int WorldW = PanelW;
	const int WorldH = PanelH;
	const int WorldSize = WorldH * WorldW;
	bool *CAGrid = (bool *)calloc(WorldSize , sizeof(bool));
	bool *next_CAGrid = (bool *)calloc(WorldSize, sizeof(bool));
	const int reqGens = 1000;
	
	//IV
	for (int i = (WorldH/2)*WorldW; i < WorldSize; i++)
	{
		CAGrid[i] = rand() % 2;
#ifdef CPUGRAPHICS
		if (i % WorldH == 0)
		{
			printf("\n");
		}
		printf("%d", CAGrid[i]);
#endif
	}
    
	hipError_t cudaStatus= CudaCAHelper(CAGrid, next_CAGrid, WorldSize,WorldH,WorldW,reqGens,&argc,argv);


	//glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_CONTINUE_EXECUTION);
	
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaCAHelper failed!");
        return 1;
    }
#ifdef CPUGRAPHICS
		for (int i = 0; i < WorldSize; i++)
		{
			if (i % WorldH == 0)
			{
				printf("\n");
			}
			printf("%d", CAGrid[i]);

		}
#endif
		printf("----------------------------");
		//system("CLS");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to do its magic 
hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size,unsigned int WorldH,unsigned int WorldW,unsigned int gen,int*argc,char**argv)
{
    hipError_t cudaStatus;
	dim3 kernelwsize(WorldW, WorldH);
	dim3 kernelbsize(1);
	int *d_WorldH=0;
	int *d_WorldW=0;
	hipDeviceProp_t myCUDA;
	if (hipGetDeviceProperties(&myCUDA, 0) == hipSuccess)
	{
		printf("Using device %d:\n", 0);
		printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
			myCUDA.name, (int)myCUDA.totalGlobalMem, (int)myCUDA.major,
			(int)myCUDA.minor, (int)myCUDA.clockRate);
		printf("Max Threads %d", myCUDA.maxThreadsPerBlock);
	}
	//int threadsPerBlock = myCUDA.maxThreadsPerBlock;
	//int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    // Choose which GPU to run on, change this on a multi-GPU system.
	
	initGLUT(argc, argv, WorldW, WorldH);
	gluOrtho2D(0, WorldW, WorldH, 0); // VIewport
	glutKeyboardFunc(keyboard);
	glutSpecialFunc(handleSpecialKeypress);
	glutPassiveMotionFunc(mouseMove);
	glutMotionFunc(mouseDrag);
	glutDisplayFunc(displayfunc);	//Display function set
	OpenGLHelper(WorldW, WorldH);	//Texture and Buffer bind
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	/*bool *d_CAGrid; */cudaStatus = hipMalloc((void**)&d_CAGrid, sizeof(bool) *size); // ALLOCATE THE SAME MEMORY SIZE AS CPU FOR GPU 

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed with Grid!",cudaStatus);
        goto Error;
    }

	bool *d_next_CAGrid; cudaStatus = hipMalloc((void**)&d_next_CAGrid, sizeof(bool) *size); // ALLOCATE THE SAME MEMORY SIZE AS CPU FOR GPU
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed with nextGrid!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
	//cudaStatus = hipMemcpy(d_WorldH, &WorldH,  sizeof(int), hipMemcpyHostToDevice);
    //cudaStatus = hipMemcpy(d_WorldW, &WorldW,  sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_CAGrid, CAGrid, size*sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	for (int i = 0; i < gen; i++)
	{
		NextGenKernel << <kernelwsize, kernelbsize >> > (d_CAGrid, d_next_CAGrid, WorldH, WorldW);
		//NextDumbKernel << <kernelwsize, kernelbsize  >> > (d_CAGrid, d_next_CAGrid);//,d_WorldH,d_WorldW);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NextGenKernel Start Error: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		std::swap(d_CAGrid, d_next_CAGrid);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching NextGenKernel!\n", cudaStatus);
			goto Error;
		}

		//Display Life Kernel Resource Mapping
		hipGraphicsMapResources(1, &cudaPboResource, 0);
		size_t num_bytes;
		cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&GLout,
			&num_bytes, cudaPboResource);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Resource Mapping Error: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		//cudaGLSetGLDevice(0);
		GLKernel << < kernelwsize, kernelbsize >> > (GLout, d_CAGrid, WorldH, WorldW);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching NextGenKernel!\n", cudaStatus);
			goto Error;
		}
		hipGraphicsUnmapResources(1, &cudaPboResource, 0);
		drawTexture(WorldW, WorldH);
		glutSwapBuffers();
		

		
	}
	glutMainLoop();

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(CAGrid, d_CAGrid, size * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    
	}

Error:
    hipFree(d_CAGrid);
    hipFree(d_next_CAGrid);
	hipGraphicsUnregisterResource(cudaPboResource);
	glDeleteBuffers(1, &GLbufferID);
	glDeleteTextures(1, &GLtexture);
    return cudaStatus;
}
