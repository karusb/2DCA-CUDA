#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>  
#include <stdio.h>
#include <algorithm>
#include <time.h>
#include <string>
#include <sstream>
#include <GL\glew.h>
#include <GL\freeglut.h>
#include "cuda_gl_interop.h"


// MANUALLY EDIT BELOW 
#define PanelW 500		// Texture Width
#define PanelH 500		// Texture Height
#define GENS 500		// number of generations to time 
#define tcompstart false	 // starts timing CPU before using GPU for given GENS
//#define ZeroBoundary		//if defined its only zero boundary at the 4 edges that defines the viewport
//



#include "Header.h"
#define nCPUGRAPHICS
#define HEURISTICS


//BEGIN GLOBAL VAR
hipGraphicsResource* cudaPboResource = nullptr;
GLuint GLtexture;
GLuint GLbufferID;
uchar4 *d_texturedata = nullptr;
uchar4 *d_bufferdata = nullptr;
uchar4 *GLout = nullptr;
bool *d_CAGrid = nullptr;
bool *d_next_CAGrid = nullptr;
bool *tempgrid = nullptr;
int evolution_number = 0;
float totalGPUtime = 0.0;
//
hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size, unsigned int WorldH, unsigned int WorldW,unsigned int gen,int*argc,char**argv);
/*
__device__ int NeighboursEval(bool *CAGrid, int x, int y,int WorldH,int WorldW)
{

	const unsigned int pos = (y*WorldW) + x;
	const unsigned int colup = x + ( (y - 1)*WorldW);
	const unsigned int coldwn = x + ((y + 1)*WorldW);
	if (x >= 0 && y >= 0 && x <= WorldW && y <= WorldH)
	{
		return  CAGrid[pos + 1] +
		 CAGrid[pos - 1]+
		 CAGrid[colup - 1]+
		 CAGrid[colup]+
		 CAGrid[colup + 1]+
		 CAGrid[coldwn - 1]+
		 CAGrid[coldwn]+
		 CAGrid[coldwn + 1];
	}
	return 0;
}
__device__ int NeighboursEval_Global(bool *CAGrid, int GlobalID, int WorldH, int WorldW)
{

	unsigned int colup = GlobalID - ((blockIdx.y - 1)*blockDim.x);
	unsigned int coldwn = GlobalID + ((blockIdx.y + 1)*blockDim.x);
	if (GlobalID > 0 && (GlobalID < WorldH * WorldW) && (GlobalID < blockDim.y * blockDim.x))
	{
		return  CAGrid[GlobalID + 1] +
			CAGrid[GlobalID - 1] +
			CAGrid[colup - 1] +
			CAGrid[colup] +
			CAGrid[colup + 1] +
			CAGrid[coldwn - 1] +
			CAGrid[coldwn] +
			CAGrid[coldwn + 1];
	}
	return 0;
}
__device__ int getGlobalIdx()
{
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}
__global__ void NextDumbKernel(bool *CAGrid, bool *NextCAGrid)
{
//int id = getGlobalIdx();
//int neighbours = 0;

}
*/
///	OpenGLHelper: Initialises texture buffers i
/// Inputs: width and height of the texture
void OpenGLHelper(unsigned int width,unsigned int height)
{

	glGenTextures(1, &GLtexture);
	glBindTexture(GL_TEXTURE_2D, GLtexture);
	
	// set basic parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// Create texture data (4-component unsigned byte)
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, d_texturedata);

	// Unbind the texture
	glBindTexture(GL_TEXTURE_2D, 0);

	
	glGenBuffers(1, &GLbufferID);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, GLbufferID);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * sizeof(uchar4), d_bufferdata, GL_STREAM_COPY);

	//glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	
	hipError_t result = hipGraphicsGLRegisterBuffer(&cudaPboResource, GLbufferID,
		cudaGraphicsMapFlagsWriteDiscard);


}
/// initGLUT: Initialises GLUT window
/// Inputs: Main function arguments argc,argv then window width,height
bool initGLUT(int* argc, char** argv,unsigned int width,unsigned int height) {
	glutInit(argc, argv);  // Create GL context.
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(width, height);
	glutCreateWindow("2Dimensional Cellular Automata (Conways Game of Life)");

	glewInit();

	if (!glewIsSupported("GL_VERSION_2_0")) {
		printf( "ERROR: Support for necessary OpenGL extensions missing.\n");
		return false;
	}

	glutReportErrors();
	return true;
}
/// drawTexture: Iteratively called draw function referenced to GLUT
/// Inputs: Texture width,height
void drawTexture(unsigned int width,unsigned int height) {
	//glColor3f(1.0f, 1.0f, 1.0f);

	gluOrtho2D(0, width*(zoomFactor+1), 0, height*(zoomFactor+1));
	
	//else glViewport(0, 0, width, height);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	//glPushMatrix();
	glViewport((loc.x ), (loc.y ), GLsizei(newpanelw*2*(zoomFactor)), GLsizei(newpanelh*2*(zoomFactor)));
	
	//gluPerspective(0, (width+loc.x) / (height+loc.y), GLdouble(loc.x/width), GLdouble(loc.y/height));
	//glTranslatef(loc.x, loc.y, 0);
	if (z1)
	{
		//gluOrtho2D(-(GLdouble)width * (GLdouble)zoomFactor, (GLdouble)width* (GLdouble)zoomFactor, -(GLdouble)height* (GLdouble)zoomFactor, (GLdouble)height* (GLdouble)zoomFactor);
		z2 = false;
		z1 = false;
		//glFlush();
	}
	if (z2)
	{
		z1 = false;
		z2 = false;
	}
	glScalef(zoomFactor, zoomFactor, 1); // scale the matrix
	//if(zoomFactor >= 1.8)glScalef(zoomFactor-0.8, zoomFactor-0.8, 1); // scale the matrix
	//glPopMatrix();
	//glMatrixMode(GL_MODELVIEW);
	//glLoadIdentity();
	

	glBindTexture(GL_TEXTURE_2D, GLtexture);
	//glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, GLbufferID);

	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glEnable(GL_TEXTURE_2D);
	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(0.0f, 0.0f);
	glTexCoord2f(1.0f, 0.0f);
	glVertex2f(float(width), 0.0f);
	glTexCoord2f(1.0f, 1.0f);
	glVertex2f(float(width), float(height));
	glTexCoord2f(0.0f, 1.0f);
	glVertex2f(0.0f, float(height));


	/////// BELOW CODE DRAWS STRING BUT DOESNT WORK
	// TODO: Integrate string printing with Texture 
	//	glColor3f(1.0f, 1.0f, 1.0f);
	//	glMatrixMode(GL_PROJECTION);
	//	glPushMatrix();
	//	glLoadIdentity();
	//	gluOrtho2D(0, newpanelw, 0, newpanelh);
	//
	//	glMatrixMode(GL_MODELVIEW);
	//	glPushMatrix();
	//	glLoadIdentity();
	//	std::string inf;
	//	std::stringstream strstream;
	//#ifndef ZeroBoundary
	//	inf = "Cyclic World";
	//#else
	//	inf = "Zero Boundary"
	//#endif // !ZeroBoundary
	//		strstream << inf << " Evolution Stage: " << evolution_number;
	//	std::string allinf(strstream.str());
	//	glRasterPos2f(0, 0);
	//	//glWindowPos2i(603, 304);
	//	for (int i = 0; i < allinf.size(); ++i) {
	//		glutBitmapCharacter(GLUT_BITMAP_TIMES_ROMAN_24, allinf[i]);
	//	}
	//	glPopMatrix();
	//
	//	glMatrixMode(GL_PROJECTION);
	//	glPopMatrix();
	glEnd();
	glDisable(GL_TEXTURE_2D);
	//glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glBindTexture(GL_TEXTURE_2D, 0);



	//glPopMatrix();
	//printf("%f", zoomFactor); //DEBUG PURPOSE
	//glFlush();

}
/// CPUGridInitLine: Initialises a line of cells every 2 rows
/// Inputs: world width,height pointer to CPU memory of the grid which only fills given scale (real number >=1)
void CPUGridInitLine(unsigned int WorldW, unsigned int WorldH, bool *CAGrid, unsigned int scale)
{
	//IV
	int row = 0;
	unsigned int WorldSize = WorldH*WorldW;
	for (int i = 1 /*(WorldH/2)*WorldW*/; i < WorldSize - WorldH * 2; i++) //changed boundaries from 0 to worldsize to shown
	{
		//CAGrid[i] = rand() % 2;
		row = i / WorldW;
		if ((i > WorldW * row + WorldW / scale) && (i < WorldW*row + WorldW - WorldW / scale) && (i > WorldW  * (WorldW / scale)) && i < WorldH*WorldW - (((WorldH / scale))*WorldW)) // Magic Code That Defines Boundaries 
		{
			if (row % 2 == 0)CAGrid[i] = 1;

		}
		else CAGrid[i] = 0;
#ifdef CPUGRAPHICS
		if (i % WorldH == 0)
		{
			printf("\n");
		}
		printf("%d", CAGrid[i]);
#endif
	}
}
/// CPUGridInitRand: Initialises a line of random cells every 2 rows
/// Inputs: world width,height pointer to CPU memory of the grid which only fills given scale (real number >=1)
void CPUGridInitRand(unsigned int WorldW, unsigned int WorldH, bool *CAGrid, unsigned int scale)
{
	//IV
	int row = 0;
	unsigned int WorldSize = WorldH*WorldW;
	for (int i = 1 /*(WorldH/2)*WorldW*/; i < WorldSize - WorldH * 2; i++) //changed boundaries from 0 to worldsize to shown
	{
		//CAGrid[i] = rand() % 2;
		row = i / WorldW;
		if ((i > WorldW * row + WorldW / scale) && (i < WorldW*row + WorldW - WorldW / scale) && (i > WorldW  * (WorldW / scale)) && i < WorldH*WorldW - (((WorldH / scale))*WorldW)) // Magic Code That Defines Boundaries 
		{
			if (row % 2 == 0)CAGrid[i] = rand()%2;

		}
		else CAGrid[i] = 0;
	}
}
/// CPUGridInitFullRand: Initialises the grid with random cells (uses rand)
/// Inputs: world width,height pointer to CPU memory 
void CPUGridInitFullRand(unsigned int WorldW, unsigned int WorldH, bool *CAGrid)
{
	//IV

	unsigned int WorldSize = WorldH*WorldW;
	for (int i = 0 /*(WorldH/2)*WorldW*/; i < WorldSize; i++) //changed boundaries from 0 to worldsize to shown
	{
		CAGrid[i] = rand() % 2;
	}
}
/// CPUNeighbours: Calculates neighbours and perform evolution to the grid on CPU
/// Inputs: pointer to CPU memory of the grid, pointer to CPU memory of the next grid,world width,height
void CPUNeighbours(bool *CAGrid, bool *NextCAGrid, int WorldH, int WorldW)
{
	int neighbours;

	for(int i =0; i < (WorldH) * (WorldW) ; i++)
	{
		int colup = i - WorldW;
		int coldwn = i + WorldW;
		int leftn = i - 1;
		int rightn = i + 1;
		if (colup < 0)colup = WorldW*(WorldH - 1) + i;
		if (coldwn > WorldH*WorldW)coldwn = i - WorldW*(WorldH-1);
		if (leftn < 0)leftn = i + WorldW;
		if (rightn > WorldW)rightn = i - WorldW;

		neighbours = CAGrid[rightn] +
			CAGrid[leftn] +
			CAGrid[colup - 1] +
			CAGrid[colup] +
			CAGrid[colup + 1] +
			CAGrid[coldwn - 1] +
			CAGrid[coldwn] +
			CAGrid[coldwn + 1];


		// END OF NEIGHBOUR ADDITION


		if (CAGrid[i] == 1)
		{
			if (neighbours < 2)
			{
				NextCAGrid[i] = 0;
			}
			else if (neighbours > 3)
			{
				NextCAGrid[i] = 0;
			}
			else if (neighbours == 3 || neighbours == 2)
			{
				NextCAGrid[i] = 1;
			}
		}
		else
		{
			if (neighbours == 3) NextCAGrid[i] = 1;
		}
	}
	std::swap(CAGrid, NextCAGrid);
}
/// CPUInsertGPU: Inserts a cell for the given location on the CPU and passes to GPU
/// Inputs: world width,height,window position of the chosen cell int2(x,y), pointer to CPU memory of the grid
/// IMPORTANT TODO : Location of the inserted cell is not accurate need to implement correct location scaling from window to texture
void CPUInsertGPU(unsigned int WorldW,unsigned int WorldH ,int2 i_loc, bool *CAGrid)
{
	float scalew = newpanelw / WorldW;
	float scaleh = newpanelh / WorldH;
	int myid = 0;
	if (scalew > 2.0 && scaleh > 2.0)
		myid = (i_loc.x / float(newpanelw / WorldW)) + (i_loc.y / float(newpanelh / WorldH))*WorldW;
	else
		myid = i_loc.x + i_loc.y * newpanelw;
	if(myid <= WorldW*WorldH && myid>0)CAGrid[myid] = !CAGrid[myid];
}
/// CUDA : NextGenKernel : Calculates the neighbours of each cell and puts the new state of the cell in NextCAGrid
/// Inputs : Pointer to the current CAGrid, pointer to the nextCAGrid, world width,height.
__global__ void NextGenKernel(bool *CAGrid, bool *NextCAGrid,int WorldH,int WorldW)
{
	//int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y*blockDim.y + threadIdx.y)* WorldH;
	int id = (blockIdx.y * gridDim.x + blockIdx.x); // GLOBAL CELL ID 
	int neighbours = 0; 
	//unsigned int WorldS = WorldH*WorldW; // World Size
	
	unsigned int colup = ((blockIdx.y - 1) * gridDim.x + blockIdx.x); //Upper Row of the Cell	 (block)
	unsigned int coldwn = ((blockIdx.y + 1) * gridDim.x +blockIdx.x); //Lower Row of the Cell (block)
#ifndef ZeroBoundary
	if (blockIdx.y == 0) colup = ((gridDim.y - 1) * gridDim.x + blockIdx.x); //Change Upper Row to lowest Row if the cell is on the first row
	if (blockIdx.y == gridDim.y-1) coldwn = blockIdx.x; // Change Lower Row to first row if the cell is on the last row
#endif

	// NON Working Mapping
	//unsigned int colup = id - ((blockIdx.y - gridDim.x)*gridDim.x);
	//unsigned int coldwn = id + ((blockIdx.y + gridDim.x)*gridDim.x);
	//unsigned int colup = ((id - (id%WorldW)) + WorldS - WorldW) % WorldS;
	//unsigned int coldwn = ((id - (id%WorldW)) + WorldW) % WorldS;
	//unsigned int colup = id - ((blockIdx.y - 1)*blockDim.x);
	//unsigned int coldwn = id + ((blockIdx.y + 1)*blockDim.x);
	

	if (id < (WorldH) * (WorldW) && id>=0 ) // Are we within the boundaries ? 
	{
		//neighbours = NeighboursEval_Global(CAGrid, id, WorldH, WorldW);
		
#ifdef ZeroBoundary // Uses this algorithm if zero boundary is defined




		if (colup <= 0)
		{ 
			if (id - 1 <= 0)
			{
				neighbours = CAGrid[id + 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];

			}
			else if (id + 1 > PanelW)
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];
			}
			else
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[id + 1] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];
			}
		}

		else if (coldwn > PanelH*PanelW)
		{
			if (id - 1 <= 0)
			{
				neighbours = CAGrid[id + 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
			else if (id + 1 > PanelW*PanelH)
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
			else
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[id + 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
		}

		//else if (id - 1 <= 0)
		//{
		//	neighbours = CAGrid[id + 1] +
		//		
		//		CAGrid[colup] +
		//		CAGrid[colup + 1] +
		//		
		//		CAGrid[coldwn] +
		//		CAGrid[coldwn + 1];
		//}
		//else if (id + 1 > PanelW)
		//{
		//	neighbours = CAGrid[id - 1] +
		//		CAGrid[colup - 1] +
		//		CAGrid[colup] +

		//		CAGrid[coldwn - 1] +
		//		CAGrid[coldwn];
		//		
		//}
		else
		{

			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];

		}
		// END OF NEIGHBOUR ADDITION
#else // Cyclic Algorithm
		if (id == 0) // First block
		{
				neighbours = CAGrid[id + 1] +
					CAGrid[gridDim.x - 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup -gridDim.x] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];

		}
		else if (id == gridDim.x*gridDim.y - 1) // Last Block
		{

			neighbours = CAGrid[gridDim.x*gridDim.y -gridDim.x] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x*gridDim.y - gridDim.x) // Last Row First Block
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[gridDim.x-1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x - 1)	//First Row Last Block
		{
			neighbours = CAGrid[id + 1]; +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup - gridDim.x + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x)	// Second Row First Block
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[gridDim.x - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
#endif // !ZeroBoundary

		if (CAGrid[id] == 1)
		{
			if (neighbours < 2)
			{
				NextCAGrid[id] = 0;
			}
			else if (neighbours > 3)
			{
				NextCAGrid[id] = 0;
			}
			else if (neighbours == 3 || neighbours == 2)
			{
				NextCAGrid[id] = 1;
			}
		}
		else
		{
			NextCAGrid[id] = 0;
			if (neighbours == 3) NextCAGrid[id] = 1;
			
		}
	}
}
/// CUDA : GLKernel : Sets the colour of the texture buffer given the correct inputs
/// Inputs: Pointer to the mapped texture , pointer to the current CAGrid, pointer to the next CAGrid, world width,height,2 state or 4 state choice
__global__ void GLKernel(uchar4 *d_buf,bool *CAGrid,bool *NextCAGrid, int WorldH, int WorldW,bool d_lifecontrol)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x);

	if (id < (WorldH) * (WorldW) && id >= 0)
	{
		if (d_lifecontrol)
		{

			if (CAGrid[id] == 1)
			{
				d_buf[id].w = 255;
				d_buf[id].x = 255;
				d_buf[id].y = 255;
				d_buf[id].z = 255;
			}
			if (CAGrid[id] == 0)
			{
				d_buf[id].w = 255;
				d_buf[id].x = 0;
				d_buf[id].y = 0;
				d_buf[id].z = 0;
			}
		}

		else
		{

			//d_buf[id].w = 255;
			////d_buf[id].x = 191;
			////d_buf[id].y = 173;
			//d_buf[id].x = 120;
			//d_buf[id].y = 113;
			//d_buf[id].z = 134;

			if (CAGrid[id] == 1 && NextCAGrid[id] == 1) //GETTING OLDER
			{
				d_buf[id].w = 255;
				d_buf[id].x = 22;
				d_buf[id].y = 78;
				d_buf[id].z = 146;
			}
			if (CAGrid[id] == 1 && NextCAGrid[id] == 0) // NEW BORN
			{
				d_buf[id].w = 255;
				d_buf[id].x = 255;
				d_buf[id].y = 255;
				d_buf[id].z = 255;
			}
			//if (CAGrid[id] == 0)	//DEAD
			//{
			//	d_buf[id].w = 255;
			//	d_buf[id].x -= 191;
			//	d_buf[id].y -= 173;
			//	d_buf[id].z -= 134;
			//}
			if (CAGrid[id] == 0 && NextCAGrid[id] == 1) // WAS ALIVE
			{
				d_buf[id].w = 255;
				d_buf[id].x = 146;
				d_buf[id].y = 22;
				d_buf[id].z = 129;
			}
			if (CAGrid[id] == 0 && NextCAGrid[id] == 0) // NO ONE
			{
				d_buf[id].w = 255;
				d_buf[id].x = 0;
				d_buf[id].y = 0;
				d_buf[id].z = 0;
			}

		}
	}
	
}
/// GLUT : displayfunc : Called everytime when  screen update is called through GLUT, runs majority of the code as well as the CUDA kernels,
/// All of the global control variables are checked here
void displayfunc()
{
	int WorldW = PanelW;
	int WorldH = PanelH;
	dim3 kernelwsize(WorldW, WorldH);
	
	dim3 kernelbsize(1);
	hipEvent_t start, stop; //CUDA timing var
	float ms = 0.0;
	glClear(GL_COLOR_BUFFER_BIT); //Clear color buf
	if (timecompare && evolution_number <= GENS)
	{
		cont = true;
		evolutioncontrol = true;
		
	}
	else if (timecompare && evolution_number >GENS)
	{
		cont = false;
		evolutioncontrol = false;
		printf("Total GPU Time %f ms \n", totalGPUtime);
		timecompare = false;
	}


#ifdef HEURISTICS
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
#endif
		if(cont)NextGenKernel << <kernelwsize, kernelbsize >> > (d_CAGrid, d_next_CAGrid, WorldH, WorldW); // NextGeneration Kernel adds neighbours and sets nextCA grid
		//NextDumbKernel << <kernelwsize, kernelbsize  >> > (d_CAGrid, d_next_CAGrid);//,d_WorldH,d_WorldW);
		// Check for any errors launching the kernel

#ifdef HEURISTICS
		//hipDeviceSynchronize();
		
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ms, start, stop);
		totalGPUtime += ms;
		if(!timecompare)printf(" Elapsed GPU Time: %f ms \n", ms);
#endif
		hipDeviceSynchronize();
		if(cont)std::swap(d_CAGrid, d_next_CAGrid); //Swaps the values of both pointers
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		if (resetlife || fullresetlife)
		{
			bool *tempgrid = (bool *)calloc(WorldH*WorldW, sizeof(bool));
			if(resetlife)CPUGridInitRand(WorldW, WorldH, tempgrid, 5);
			if(fullresetlife)CPUGridInitFullRand(WorldW, WorldH, tempgrid);
			hipMemset(d_CAGrid, 0, sizeof(d_CAGrid)); // reset the current grid
			if (hipMemcpy(d_CAGrid, tempgrid, WorldH*WorldW * sizeof(bool), hipMemcpyHostToDevice) != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
			hipMemset(d_next_CAGrid, 0, sizeof(d_next_CAGrid)); // reset the next grid
			free(tempgrid);
			resetlife = false;
			fullresetlife = false;
		}
		if (givelife)
		{
			bool *tempgrid = (bool *)calloc(WorldH*WorldW, sizeof(bool));
			if (hipMemcpy(tempgrid, d_CAGrid, WorldH*WorldW * sizeof(bool), hipMemcpyDeviceToHost) != hipSuccess) {
				fprintf(stderr, "hipMemcpy at display function for Device to Host failed!");
			}

			CPUInsertGPU(WorldW,WorldH, loc2, tempgrid);

			if (hipMemcpy(d_CAGrid, tempgrid, WorldH*WorldW * sizeof(bool), hipMemcpyHostToDevice) != hipSuccess) {
				fprintf(stderr, "hipMemcpy at display function for Host to Device failed!");
			}
			givelife = false;
		}
		if(cont)evolution_number += 1;
		if(!timecompare)printf("Evolution Stage %d", evolution_number);

	



	hipGraphicsMapResources(1, &cudaPboResource, 0); // map memory
	size_t num_bytes;
	hipError_t cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&GLout, // map to pointed texture
		&num_bytes, cudaPboResource);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Resource Mapping Error: %s\n", hipGetErrorString(cudaStatus));
		}
	//cudaGLSetGLDevice(0);
	GLKernel << < kernelwsize, kernelbsize >> > (GLout, d_CAGrid,d_next_CAGrid, WorldH, WorldW,lifecontrol); //Fills GL texture with CA data
																											 
	hipGraphicsUnmapResources(1, &cudaPboResource, 0); //unmap resource memory
	drawTexture(WorldW, WorldH); // call texture draw function




	glutSwapBuffers(); //swap back buffer with front buffer

	//hipMemset(d_next_CAGrid, 0, sizeof(d_next_CAGrid)); // reset the next grid
	glFlush();
	if (evolutioncontrol)glutPostRedisplay(); //for consecutive frame update hence evolution if set
	//glutPostRedisplay();


}
int main(int argc,char** argv)
{
    

	const int WorldW = PanelW;
	const int WorldH = PanelH;
	const int WorldSize = WorldH * WorldW;
	bool *CAGrid = (bool *)calloc(WorldSize , sizeof(bool)); // Allocate world 
	bool *next_CAGrid = (bool *)calloc(WorldSize, sizeof(bool));
	
	const int reqGens = GENS;
	// BEGIN INFO
	printf("Starting GLUT main loop...\n");
	printf("Press [r] to reset the view to a randomized board \n")  ;
	printf("Press [f] to reset the view to a fully randomized board \n");
	printf("Press [ESC] to exit \n" ) ;
	printf( "Press the [+] key to zoom in \n")  ;
	printf( "Press the [-] key to zoom out \n")  ;
	printf( "Press the [up arrow] to move up \n")  ;
	printf( "Press the [down arrow] to move down \n")  ;
	printf( "Press the [left arrow] to move left \n")  ;
	printf( "Press the [right arrow] to move right \n")  ;
	printf( "Press the [l] key to switch between colour and colourless \n")  ;
	printf( "Press the [space] bar to stop evolution \n")  ;
	printf("Press the [e] key to evolve consecutively \n");
	printf("Press the [d] key to activate mouse functions \n");
	printf("MouseWheelUp = zoom+ - MouseWheelDown = zoom- \n");
	printf("MouseLeftClick and Drag to change viewing position \n");
	printf("MouseRightClick = Spawn or Kill a cell at the mouse location \n");
	printf("Press the [t] key to activate timing mode for GPU for %d generations \n",GENS);


///// BEGIN GRID INIT
	CPUGridInitLine(WorldW, WorldH, CAGrid, 5);

//////
// BEGIN CPU NEIGHBOUR CALCULATION
#ifdef HEURISTICS
	clock_t start = clock(), diff;
	if(tcompstart)for (int k = 0;k < reqGens ; k++)CPUNeighbours(CAGrid, next_CAGrid, WorldH, WorldW);
	else CPUNeighbours(CAGrid, next_CAGrid, WorldH, WorldW);
	diff = clock() - start;
	int msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("Time taken %d seconds %d milliseconds for CPU", msec / 1000, msec % 1000);
	if (tcompstart)CPUGridInitLine(WorldW, WorldH, CAGrid, 5);
#endif
	
	// CUDA&GLUT Initialise Function
	hipError_t cudaStatus= CudaCAHelper(CAGrid, next_CAGrid, WorldSize,WorldH,WorldW,reqGens,&argc,argv);
	

	//glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_CONTINUE_EXECUTION);
	
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaCAHelper failed!");
        return 1;
    }
#ifdef CPUGRAPHICS
		for (int i = 0; i < WorldSize; i++)
		{
			if (i % WorldH == 0)
			{
				printf("\n");
			}
			printf("%d", CAGrid[i]);

		}
#endif
		printf("----------------------------");
		//system("CLS");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

/// CudaCAHelper : Allocates Memory for CAGrid and NextCAGrid on GPU for given size
/// Inputs : Pointer to current CAGrid, pointer to NextCAGrid, world size (w*h), world height,width, required generations
hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size,unsigned int WorldH,unsigned int WorldW,unsigned int gen,int*argc,char**argv)
{
    hipError_t cudaStatus;
	dim3 kernelwsize(WorldW, WorldH);
	dim3 kernelbsize(1);

	hipDeviceProp_t myCUDA;
	if (hipGetDeviceProperties(&myCUDA, 0) == hipSuccess)
	{
		printf("Using device %d:\n", 0);
		printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
			myCUDA.name, (int)myCUDA.totalGlobalMem, (int)myCUDA.major,
			(int)myCUDA.minor, (int)myCUDA.clockRate);
		printf("Max Threads %d", myCUDA.maxThreadsPerBlock);
	}
	//int threadsPerBlock = myCUDA.maxThreadsPerBlock;
	//int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
	// Allocate GPU buffers for three vectors (two input, one output)    .
	/*bool *d_CAGrid; */cudaStatus = hipMalloc((void**)&d_CAGrid, sizeof(bool) *size); // ALLOCATE THE SAME MEMORY SIZE AS CPU FOR GPU 

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed with Grid! %d", cudaStatus);
		goto Error;
	}

	/*bool *d_next_CAGrid; */cudaStatus = hipMalloc((void**)&d_next_CAGrid, sizeof(bool) *size); // ALLOCATE THE SAME MEMORY SIZE AS CPU FOR GPU
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed with nextGrid!");
		goto Error;
	}

	/// START GPU TIMING 
	if (tcompstart)
	{
		hipEvent_t startgpu, stopgpu; //CUDA timing var
		float mstwo = 0.0;
		hipEventCreate(&startgpu);
		hipEventCreate(&stopgpu);
		hipEventRecord(startgpu, 0);
		for (int i = 0; i < GENS; i++)NextGenKernel << <kernelwsize, kernelbsize >> > (d_CAGrid, d_next_CAGrid, WorldH, WorldW); // NextGeneration Kernel adds neighbours and sets nextCA grid
		hipEventRecord(stopgpu, 0);
		hipEventSynchronize(stopgpu);
		hipEventElapsedTime(&mstwo, startgpu, stopgpu);
		printf(" Elapsed GPU Time: %f ms \n", mstwo);
		hipMemset(d_CAGrid, 0, sizeof(d_CAGrid));
		hipMemset(d_next_CAGrid, 0, sizeof(d_next_CAGrid));
		CPUGridInitLine(WorldW, WorldH, CAGrid, 5);
	}
	///

	initGLUT(argc, argv, WorldW, WorldH);
	gluOrtho2D(0, WorldW, WorldH, 0); // VIewport
	glutKeyboardFunc(keyboard);		//keyboard press func
	glutSpecialFunc(handleSpecialKeypress); //arrow keys
	glutMouseFunc(mouseCall);		// mouse clicks
	glutMotionFunc(mouseMove);		// mouse motion
	glutReshapeFunc(reshape);		// windows reshape function
	glutDisplayFunc(displayfunc);	//Display function set
	OpenGLHelper(WorldW, WorldH);	//Texture and Buffer bind
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }




    // Copy input vectors from host memory to GPU buffers.
	//cudaStatus = hipMemcpy(d_WorldH, &WorldH,  sizeof(int), hipMemcpyHostToDevice);
    //cudaStatus = hipMemcpy(d_WorldW, &WorldW,  sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_CAGrid, CAGrid, size*sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


	// BEGIN MAIN GLUT LOOP
	glutMainLoop();
	//
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(CAGrid, d_CAGrid, size * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    
	}
	// BEGIN FREE ALLOCATED MEMORY
	free(CAGrid);
	free(tempgrid);
	free(NextCAGrid);
	hipFree(d_CAGrid);
	hipFree(d_next_CAGrid);
	hipGraphicsUnregisterResource(cudaPboResource);
	glDeleteBuffers(1, &GLbufferID);
	glDeleteTextures(1, &GLtexture);
Error:
    hipFree(d_CAGrid);
    hipFree(d_next_CAGrid);
	hipGraphicsUnregisterResource(cudaPboResource);
	glDeleteBuffers(1, &GLbufferID);
	glDeleteTextures(1, &GLtexture);
    return cudaStatus;
	//
}
