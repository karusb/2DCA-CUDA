#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>  
#include <stdio.h>
#include <algorithm>
#include <time.h>
#include <string>
#include <sstream>

#define PanelW 1000
#define PanelH 1000
//#define ZeroBoundary
//

//#include <GL\GL.h>
//#include <GL\GLU.h>
//#include <GL\glut.h>
//#include <cudaGL.h>
#include <GL\glew.h>
//#include <GL\glxew.h>
//#include <GL\wglew.h>
#include <GL\freeglut.h>
#include "cuda_gl_interop.h"
#include "Header.h"
#define nCPUGRAPHICS
#define HEURISTICS



hipGraphicsResource* cudaPboResource = nullptr;
GLuint GLtexture;
GLuint GLbufferID;
uchar4 *d_texturedata = nullptr;
uchar4 *d_bufferdata = nullptr;
uchar4 *GLout = nullptr;
bool *d_CAGrid = nullptr;
bool *d_next_CAGrid = nullptr;
int evolution_number = 0;

hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size, unsigned int WorldH, unsigned int WorldW,unsigned int gen,int*argc,char**argv);
/*
__device__ int NeighboursEval(bool *CAGrid, int x, int y,int WorldH,int WorldW)
{

	const unsigned int pos = (y*WorldW) + x;
	const unsigned int colup = x + ( (y - 1)*WorldW);
	const unsigned int coldwn = x + ((y + 1)*WorldW);
	if (x >= 0 && y >= 0 && x <= WorldW && y <= WorldH)
	{
		return  CAGrid[pos + 1] +
		 CAGrid[pos - 1]+
		 CAGrid[colup - 1]+
		 CAGrid[colup]+
		 CAGrid[colup + 1]+
		 CAGrid[coldwn - 1]+
		 CAGrid[coldwn]+
		 CAGrid[coldwn + 1];
	}
	return 0;
}
__device__ int NeighboursEval_Global(bool *CAGrid, int GlobalID, int WorldH, int WorldW)
{

	unsigned int colup = GlobalID - ((blockIdx.y - 1)*blockDim.x);
	unsigned int coldwn = GlobalID + ((blockIdx.y + 1)*blockDim.x);
	if (GlobalID > 0 && (GlobalID < WorldH * WorldW) && (GlobalID < blockDim.y * blockDim.x))
	{
		return  CAGrid[GlobalID + 1] +
			CAGrid[GlobalID - 1] +
			CAGrid[colup - 1] +
			CAGrid[colup] +
			CAGrid[colup + 1] +
			CAGrid[coldwn - 1] +
			CAGrid[coldwn] +
			CAGrid[coldwn + 1];
	}
	return 0;
}
__device__ int getGlobalIdx()
{
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}
__global__ void NextDumbKernel(bool *CAGrid, bool *NextCAGrid)
{
//int id = getGlobalIdx();
//int neighbours = 0;

}
*/

void OpenGLHelper(unsigned int width,unsigned int height)
{

	glGenTextures(1, &GLtexture);
	glBindTexture(GL_TEXTURE_2D, GLtexture);
	
	// set basic parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// Create texture data (4-component unsigned byte)
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, d_texturedata);

	// Unbind the texture
	glBindTexture(GL_TEXTURE_2D, 0);

	
	glGenBuffers(1, &GLbufferID);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, GLbufferID);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * sizeof(uchar4), d_bufferdata, GL_STREAM_COPY);

	//glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	
	hipError_t result = hipGraphicsGLRegisterBuffer(&cudaPboResource, GLbufferID,
		cudaGraphicsMapFlagsWriteDiscard);


}
bool initGLUT(int* argc, char** argv,unsigned int width,unsigned int height) {
	glutInit(argc, argv);  // Create GL context.
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(width, height);
	glutCreateWindow("2Dimensional Cellular Automata (Conways Game of Life)");

	glewInit();

	if (!glewIsSupported("GL_VERSION_2_0")) {
		printf( "ERROR: Support for necessary OpenGL extensions missing.\n");
		return false;
	}

	glutReportErrors();
	return true;
}
void drawTexture(unsigned int width,unsigned int height) {
	//glColor3f(1.0f, 1.0f, 1.0f);

	glViewport(loc.x / 2, loc.y / 2, width, height);
	//else glViewport(0, 0, width, height);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	//glPushMatrix();

	if (z1)
	{
		//gluOrtho2D(-(GLdouble)width * (GLdouble)zoomFactor, (GLdouble)width* (GLdouble)zoomFactor, -(GLdouble)height* (GLdouble)zoomFactor, (GLdouble)height* (GLdouble)zoomFactor);
		z2 = false;
		z1 = false;
		//glFlush();
	}
	if (z2)
	{
		z1 = false;
		z2 = false;
	}
	glScalef(zoomFactor, zoomFactor, 1); // scale the matrix

	//glPopMatrix();
	//glMatrixMode(GL_MODELVIEW);
	//glLoadIdentity();
	

	glBindTexture(GL_TEXTURE_2D, GLtexture);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, GLbufferID);

	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glEnable(GL_TEXTURE_2D);
	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(0.0f, 0.0f);
	glTexCoord2f(1.0f, 0.0f);
	glVertex2f(float(width), 0.0f);
	glTexCoord2f(1.0f, 1.0f);
	glVertex2f(float(width), float(height));
	glTexCoord2f(0.0f, 1.0f);
	glVertex2f(0.0f, float(height));
	glEnd();
	glDisable(GL_TEXTURE_2D);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glBindTexture(GL_TEXTURE_2D, 0);



	//glPopMatrix();
	//glFlush();

}
void reshape(int w, int h)
{
	glViewport(0.0, 0.0, (GLsizei)w, (GLsizei)h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(-(GLdouble)w * zoomFactor, (GLdouble)w* zoomFactor, -(GLdouble)h* zoomFactor, (GLdouble)h* zoomFactor);
}
void CPUNeighbours(bool *CAGrid, bool *NextCAGrid, int WorldH, int WorldW)
{
	int neighbours;

	for(int i =0; i < (WorldH) * (WorldW) ; i++)
	{
		int colup = i - WorldW;
		int coldwn = i + WorldW;
		int leftn = i - 1;
		int rightn = i + 1;
		if (colup < 0)colup = WorldW*(WorldH - 1) + i;
		if (coldwn > WorldH*WorldW)coldwn = i - WorldW*(WorldH-1);
		if (leftn < 0)leftn = i + WorldW;
		if (rightn > WorldW)rightn = i - WorldW;

		neighbours = CAGrid[rightn] +
			CAGrid[leftn] +
			CAGrid[colup - 1] +
			CAGrid[colup] +
			CAGrid[colup + 1] +
			CAGrid[coldwn - 1] +
			CAGrid[coldwn] +
			CAGrid[coldwn + 1];


		// END OF NEIGHBOUR ADDITION



		if (neighbours < 2)
		{
			NextCAGrid[i] = 0;
		}
		else if (neighbours > 3)
		{
			NextCAGrid[i] = 0;
		}
		else if (neighbours == 3)
		{
			NextCAGrid[i] = 1;
		}
	}
}

__global__ void NextGenKernel(bool *CAGrid, bool *NextCAGrid,int WorldH,int WorldW)
{
	//int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y*blockDim.y + threadIdx.y)* WorldH;
	int id = (blockIdx.y * gridDim.x + blockIdx.x); // GLOBAL CELL ID 
	int neighbours = 0; 
	unsigned int WorldS = WorldH*WorldW; // World Size
	
	unsigned int colup = ((blockIdx.y - 1) * gridDim.x + blockIdx.x); //Upper Row of the Cell	 (block)
	unsigned int coldwn = ((blockIdx.y + 1) * gridDim.x +blockIdx.x); //Lower Row of the Cell (block)
#ifndef ZeroBoundary
	if (blockIdx.y == 0) colup = ((gridDim.y - 1) * gridDim.x + blockIdx.x); //Change Upper Row to lowest Row if the cell is on the first row
	if (blockIdx.y == gridDim.y-1) coldwn = blockIdx.x; // Change Lower Row to first row if the cell is on the last row
#endif

	// NON Working Mapping
	//unsigned int colup = id - ((blockIdx.y - gridDim.x)*gridDim.x);
	//unsigned int coldwn = id + ((blockIdx.y + gridDim.x)*gridDim.x);
	//unsigned int colup = ((id - (id%WorldW)) + WorldS - WorldW) % WorldS;
	//unsigned int coldwn = ((id - (id%WorldW)) + WorldW) % WorldS;
	//unsigned int colup = id - ((blockIdx.y - 1)*blockDim.x);
	//unsigned int coldwn = id + ((blockIdx.y + 1)*blockDim.x);
	if (id < (WorldH) * (WorldW) && id>=0 ) // Are we within the boundaries ? 
	{
		//neighbours = NeighboursEval_Global(CAGrid, id, WorldH, WorldW);

#ifdef ZeroBoundary // Uses this algorithm if zero boundary is defined




		if (colup <= 0)
		{ 
			if (id - 1 <= 0)
			{
				neighbours = CAGrid[id + 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];

			}
			else if (id + 1 > PanelW)
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];
			}
			else
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[id + 1] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];
			}
		}

		else if (coldwn > PanelH*PanelW)
		{
			if (id - 1 <= 0)
			{
				neighbours = CAGrid[id + 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
			else if (id + 1 > PanelW*PanelH)
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
			else
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[id + 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
		}

		//else if (id - 1 <= 0)
		//{
		//	neighbours = CAGrid[id + 1] +
		//		
		//		CAGrid[colup] +
		//		CAGrid[colup + 1] +
		//		
		//		CAGrid[coldwn] +
		//		CAGrid[coldwn + 1];
		//}
		//else if (id + 1 > PanelW)
		//{
		//	neighbours = CAGrid[id - 1] +
		//		CAGrid[colup - 1] +
		//		CAGrid[colup] +

		//		CAGrid[coldwn - 1] +
		//		CAGrid[coldwn];
		//		
		//}
		else
		{

			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];

		}
		// END OF NEIGHBOUR ADDITION
#else // Cyclic Algorithm
		if (id == 0) // First block?
		{
				neighbours = CAGrid[id + 1] +
					CAGrid[gridDim.x - 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup -gridDim.x] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];

		}
		else if (id == gridDim.x*gridDim.y - 1) // Last Block
		{

			neighbours = CAGrid[gridDim.x*gridDim.y -gridDim.x] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x*gridDim.y - gridDim.x) // Last Row First Block
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[gridDim.x-1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x - 1)	//First Row Last Block
		{
			neighbours = CAGrid[id + 1]; +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup - gridDim.x + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x)	// Second Row First Block
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[gridDim.x - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
#endif // !ZeroBoundary

		if (CAGrid[id] == 1)
		{
			if (neighbours < 2)
			{
				NextCAGrid[id] = 0;
			}
			else if (neighbours > 3)
			{
				NextCAGrid[id] = 0;
			}
			else if (neighbours == 3 || neighbours == 2)
			{
				NextCAGrid[id] = 1;
			}
		}
		else
		{
			if (neighbours == 3) NextCAGrid[id] = 1;
		}
	}
}
__global__ void GLKernel(uchar4 *d_buf,bool *CAGrid,bool *NextCAGrid, int WorldH, int WorldW,bool d_lifecontrol)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x);

	if (id < (WorldH) * (WorldW) && id >= 0)
	{
		if (d_lifecontrol)
		{

			if (CAGrid[id] == 1)
			{
				d_buf[id].w = 255;
				d_buf[id].x = 255;
				d_buf[id].y = 255;
				d_buf[id].z = 255;
			}
			if (CAGrid[id] == 0)
			{
				d_buf[id].w = 255;
				d_buf[id].x = 0;
				d_buf[id].y = 0;
				d_buf[id].z = 0;
			}
		}

		else
		{

			d_buf[id].w = 255;
			d_buf[id].x = 191;
			d_buf[id].y = 173;
			d_buf[id].z = 134;

			if (CAGrid[id] == 1 && NextCAGrid[id] == 1) //GETTING OLDER
			{
				d_buf[id].w = 255;
				d_buf[id].x -= 50;
				d_buf[id].y -= 50;
				d_buf[id].z -= 50;
			}
			if (CAGrid[id] == 1 && NextCAGrid[id] == 0) // NEW BORN
			{
				d_buf[id].w = 255;
				d_buf[id].x += 64;
				d_buf[id].y += 82;
				d_buf[id].z += 121;
			}
			if (CAGrid[id] == 0)	//DEAD
			{
				d_buf[id].w = 255;
				d_buf[id].x -= 191;
				d_buf[id].y -= 173;
				d_buf[id].z -= 134;
			}
			if (CAGrid[id] == 0 && NextCAGrid[id] == 1) // WAS ALIVE
			{
				d_buf[id].w = 255;
				d_buf[id].x += 50;
				d_buf[id].y += 50;
				d_buf[id].z += 50;
			}
			if (CAGrid[id] == 0 && NextCAGrid[id] == 0) // NO ONE
			{
				d_buf[id].w = 255;
				d_buf[id].x = 0;
				d_buf[id].y = 0;
				d_buf[id].z = 0;
			}

		}
	}
	
}
void displayfunc()
{
	int WorldW = PanelW;
	int WorldH = PanelH;
	dim3 kernelwsize(WorldW, WorldH);
	
	dim3 kernelbsize(1);
	hipEvent_t start, stop; //CUDA timing var
	float ms;
	glClear(GL_COLOR_BUFFER_BIT);
	if (cont)
	{
#ifdef HEURISTICS
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
#endif
		NextGenKernel << <kernelwsize, kernelbsize >> > (d_CAGrid, d_next_CAGrid, WorldH, WorldW);
		//NextDumbKernel << <kernelwsize, kernelbsize  >> > (d_CAGrid, d_next_CAGrid);//,d_WorldH,d_WorldW);
		// Check for any errors launching the kernel

#ifdef HEURISTICS
		hipDeviceSynchronize();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ms, start, stop);
		printf(" Elapsed GPU Time: %f ms \n", ms);
#endif
		hipDeviceSynchronize();
		std::swap(d_CAGrid, d_next_CAGrid);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		evolution_number += 1;
		printf("Evolution Stage %d", evolution_number);

	}



	hipGraphicsMapResources(1, &cudaPboResource, 0);
	size_t num_bytes;
	hipError_t cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&GLout,
		&num_bytes, cudaPboResource);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Resource Mapping Error: %s\n", hipGetErrorString(cudaStatus));
		}
	//cudaGLSetGLDevice(0);
	GLKernel << < kernelwsize, kernelbsize >> > (GLout, d_CAGrid,d_next_CAGrid, WorldH, WorldW,lifecontrol);
	hipGraphicsUnmapResources(1, &cudaPboResource, 0);
	drawTexture(WorldW, WorldH);
// TODO: Integrate string printing with Texture 
	/*
	std::string inf;
	std::stringstream strstream;
	#ifndef ZeroBoundary
	inf = "Cyclic World";
	#else
	inf = "Zero Boundary"
	#endif // !ZeroBoundary
	strstream << inf << " Evolution Stage: " << evolution_number;
	std::string allinf(strstream.str());
	glColor3f(1.0f, 1.0f, 1.0f);
	glRasterPos2f(-0.5f, 0.5f);
	for (int i = 0; i < allinf.size(); ++i) {
	glutBitmapCharacter(GLUT_BITMAP_TIMES_ROMAN_24, allinf[i]);
	}
	*/
	glutSwapBuffers();
	if (evolutioncontrol)glutPostRedisplay();
	//glutPostRedisplay();

}
int main(int argc,char** argv)
{
    

	const int WorldW = PanelW;
	const int WorldH = PanelH;
	const int WorldSize = WorldH * WorldW;
	bool *CAGrid = (bool *)calloc(WorldSize , sizeof(bool));
	bool *next_CAGrid = (bool *)calloc(WorldSize, sizeof(bool));
	const int reqGens = 1000;

	printf("Starting GLUT main loop...\n");
	printf("Press [r] to reset the view to the original \n")  ;
	printf("Press [ESC] to exit \n" ) ;
	printf( "Press the [+] key to zoom in \n")  ;
	printf( "Press the [-] key to zoom out \n")  ;
	printf( "Press the [up arrow] to move up \n")  ;
	printf( "Press the [down arrow] to move down \n")  ;
	printf( "Press the [left arrow] to move left \n")  ;
	printf( "Press the [right arrow] to move right \n")  ;
	printf( "Press the [l] key to render with colours \n")  ;
	printf( "Press the [space] bar to stop evolution \n")  ;
	printf("Press the [e] key to evolve consecutively \n");


	//IV
	int row = 0;
	for (int i =1 /*(WorldH/2)*WorldW*/; i < WorldSize - WorldH*2; i++) //changed boundaries from 0 to worldsize to shown
	{
		//CAGrid[i] = rand() % 2;
		row = i / WorldW;
		if ((i > WorldW * row + WorldW / 5) && (i < WorldW*row + WorldW - WorldW / 5) && (i > WorldW  * (WorldW / 5)) && i < WorldH*WorldW  - (((WorldH / 5))*WorldW)) // Magic Code That Defines Boundaries 
		{
			if (row % 2 == 0)CAGrid[i] = 1;
			
		}
		else CAGrid[i] = 0;
#ifdef CPUGRAPHICS
		if (i % WorldH == 0)
		{
			printf("\n");
		}
		printf("%d", CAGrid[i]);
#endif
	}
#ifdef HEURISTICS
	clock_t start = clock(), diff;
	CPUNeighbours(CAGrid, next_CAGrid, WorldH, WorldW);
	diff = clock() - start;
	int msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("Time taken %d seconds %d milliseconds for CPU", msec / 1000, msec % 1000);
#endif
	hipError_t cudaStatus= CudaCAHelper(CAGrid, next_CAGrid, WorldSize,WorldH,WorldW,reqGens,&argc,argv);


	//glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_CONTINUE_EXECUTION);
	
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaCAHelper failed!");
        return 1;
    }
#ifdef CPUGRAPHICS
		for (int i = 0; i < WorldSize; i++)
		{
			if (i % WorldH == 0)
			{
				printf("\n");
			}
			printf("%d", CAGrid[i]);

		}
#endif
		printf("----------------------------");
		//system("CLS");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to do its magic 
hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size,unsigned int WorldH,unsigned int WorldW,unsigned int gen,int*argc,char**argv)
{
    hipError_t cudaStatus;
	dim3 kernelwsize(WorldW, WorldH);
	dim3 kernelbsize(1);

	hipDeviceProp_t myCUDA;
	if (hipGetDeviceProperties(&myCUDA, 0) == hipSuccess)
	{
		printf("Using device %d:\n", 0);
		printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
			myCUDA.name, (int)myCUDA.totalGlobalMem, (int)myCUDA.major,
			(int)myCUDA.minor, (int)myCUDA.clockRate);
		printf("Max Threads %d", myCUDA.maxThreadsPerBlock);
	}
	//int threadsPerBlock = myCUDA.maxThreadsPerBlock;
	//int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
	
	initGLUT(argc, argv, WorldW, WorldH);
	gluOrtho2D(0, WorldW, WorldH, 0); // VIewport
	glutKeyboardFunc(keyboard);
	glutSpecialFunc(handleSpecialKeypress);
	glutMouseFunc(mouseCall);
	glutMotionFunc(mouseMove);
	glutReshapeFunc(reshape);
	glutDisplayFunc(displayfunc);	//Display function set
	OpenGLHelper(WorldW, WorldH);	//Texture and Buffer bind
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	/*bool *d_CAGrid; */cudaStatus = hipMalloc((void**)&d_CAGrid, sizeof(bool) *size); // ALLOCATE THE SAME MEMORY SIZE AS CPU FOR GPU 

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed with Grid!",cudaStatus);
        goto Error;
    }

	/*bool *d_next_CAGrid; */cudaStatus = hipMalloc((void**)&d_next_CAGrid, sizeof(bool) *size); // ALLOCATE THE SAME MEMORY SIZE AS CPU FOR GPU
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed with nextGrid!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
	//cudaStatus = hipMemcpy(d_WorldH, &WorldH,  sizeof(int), hipMemcpyHostToDevice);
    //cudaStatus = hipMemcpy(d_WorldW, &WorldW,  sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_CAGrid, CAGrid, size*sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	//REPLACED BY GLUT DISPLAY FUNC
	/*
	for (int i = 0; i < gen; i++)
	{
		NextGenKernel << <kernelwsize, kernelbsize >> > (d_CAGrid, d_next_CAGrid, WorldH, WorldW);
		//NextDumbKernel << <kernelwsize, kernelbsize  >> > (d_CAGrid, d_next_CAGrid);//,d_WorldH,d_WorldW);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NextGenKernel Start Error: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		std::swap(d_CAGrid, d_next_CAGrid);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching NextGenKernel!\n", cudaStatus);
			goto Error;
		}

		//Display Life Kernel Resource Mapping
		hipGraphicsMapResources(1, &cudaPboResource, 0);
		size_t num_bytes;
		cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&GLout,
			&num_bytes, cudaPboResource);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Resource Mapping Error: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		//cudaGLSetGLDevice(0);
		GLKernel << < kernelwsize, kernelbsize >> > (GLout, d_CAGrid, WorldH, WorldW);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching NextGenKernel!\n", cudaStatus);
			goto Error;
		}
		hipGraphicsUnmapResources(1, &cudaPboResource, 0);
		drawTexture(WorldW, WorldH);
		glutSwapBuffers();
		

		
	}
	*/
	glutMainLoop();

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(CAGrid, d_CAGrid, size * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    
	}
	hipFree(d_CAGrid);
	hipFree(d_next_CAGrid);
	hipGraphicsUnregisterResource(cudaPboResource);
	glDeleteBuffers(1, &GLbufferID);
	glDeleteTextures(1, &GLtexture);
Error:
    hipFree(d_CAGrid);
    hipFree(d_next_CAGrid);
	hipGraphicsUnregisterResource(cudaPboResource);
	glDeleteBuffers(1, &GLbufferID);
	glDeleteTextures(1, &GLtexture);
    return cudaStatus;
}
