#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>  
#include <stdio.h>
#include <algorithm>
#include <time.h>
#define PanelW 100
#define PanelH 100
//#define ZeroBoundary
//

//#include <GL\GL.h>
//#include <GL\GLU.h>
//#include <GL\glut.h>
//#include <cudaGL.h>
#include <GL\glew.h>
//#include <GL\glxew.h>
//#include <GL\wglew.h>
#include <GL\freeglut.h>
#include "cuda_gl_interop.h"
#include "Header.h"
#define nCPUGRAPHICS
#define HEURISTICS



hipGraphicsResource* cudaPboResource = nullptr;
GLuint GLtexture;
GLuint GLbufferID;
uchar4 *d_texturedata = nullptr;
uchar4 *d_bufferdata = nullptr;
uchar4 *GLout = nullptr;
bool *d_CAGrid = nullptr;
bool *d_next_CAGrid = nullptr;

hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size, unsigned int WorldH, unsigned int WorldW,unsigned int gen,int*argc,char**argv);
/*
__device__ int NeighboursEval(bool *CAGrid, int x, int y,int WorldH,int WorldW)
{

	const unsigned int pos = (y*WorldW) + x;
	const unsigned int colup = x + ( (y - 1)*WorldW);
	const unsigned int coldwn = x + ((y + 1)*WorldW);
	if (x >= 0 && y >= 0 && x <= WorldW && y <= WorldH)
	{
		return  CAGrid[pos + 1] +
		 CAGrid[pos - 1]+
		 CAGrid[colup - 1]+
		 CAGrid[colup]+
		 CAGrid[colup + 1]+
		 CAGrid[coldwn - 1]+
		 CAGrid[coldwn]+
		 CAGrid[coldwn + 1];
	}
	return 0;
}
__device__ int NeighboursEval_Global(bool *CAGrid, int GlobalID, int WorldH, int WorldW)
{

	unsigned int colup = GlobalID - ((blockIdx.y - 1)*blockDim.x);
	unsigned int coldwn = GlobalID + ((blockIdx.y + 1)*blockDim.x);
	if (GlobalID > 0 && (GlobalID < WorldH * WorldW) && (GlobalID < blockDim.y * blockDim.x))
	{
		return  CAGrid[GlobalID + 1] +
			CAGrid[GlobalID - 1] +
			CAGrid[colup - 1] +
			CAGrid[colup] +
			CAGrid[colup + 1] +
			CAGrid[coldwn - 1] +
			CAGrid[coldwn] +
			CAGrid[coldwn + 1];
	}
	return 0;
}
__device__ int getGlobalIdx()
{
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}
__global__ void NextDumbKernel(bool *CAGrid, bool *NextCAGrid)
{
//int id = getGlobalIdx();
//int neighbours = 0;

}
*/

void OpenGLHelper(unsigned int width,unsigned int height)
{

	glGenTextures(1, &GLtexture);
	glBindTexture(GL_TEXTURE_2D, GLtexture);
	
	// set basic parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// Create texture data (4-component unsigned byte)
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, d_texturedata);

	// Unbind the texture
	glBindTexture(GL_TEXTURE_2D, 0);

	
	glGenBuffers(1, &GLbufferID);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, GLbufferID);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * sizeof(uchar4), d_bufferdata, GL_STREAM_COPY);

	//glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	
	hipError_t result = hipGraphicsGLRegisterBuffer(&cudaPboResource, GLbufferID,
		cudaGraphicsMapFlagsWriteDiscard);


}
bool initGLUT(int* argc, char** argv,unsigned int width,unsigned int height) {
	glutInit(argc, argv);  // Create GL context.
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(width, height);
	glutCreateWindow("2Dimensional Cellular Automata (Conways Game of Life)");

	glewInit();

	if (!glewIsSupported("GL_VERSION_2_0")) {
		printf( "ERROR: Support for necessary OpenGL extensions missing.\n");
		return false;
	}

	glutReportErrors();
	return true;
}
void drawTexture(unsigned int width,unsigned int height) {
	//glColor3f(1.0f, 1.0f, 1.0f);
	//glMatrixMode(GL_PROJECTION);
	//glLoadIdentity();

	//glMatrixMode(GL_MODELVIEW);
	//glLoadIdentity();
	//glPushMatrix();
	if (z1)
	{
		glScalef(2, 2, 1); // scale the matrix
		z2 = false;
	}
	if (z2)
	{
		glScalef(0.5, 0.5, 1); // scale the matrix
		z1 = false;
	}
	//glPopMatrix();
	//glTranslatef(loc.x, loc.y, 0.0f);

	//

	//gluPerspective(1, (double)width / (double)height, 1.0, 300.0);
	glBindTexture(GL_TEXTURE_2D, GLtexture);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, GLbufferID);

	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glEnable(GL_TEXTURE_2D);
	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(0.0f, 0.0f);
	glTexCoord2f(1.0f, 0.0f);
	glVertex2f(float(width), 0.0f);
	glTexCoord2f(1.0f, 1.0f);
	glVertex2f(float(width), float(height));
	glTexCoord2f(0.0f, 1.0f);
	glVertex2f(0.0f, float(height));
	glEnd();
	glDisable(GL_TEXTURE_2D);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glBindTexture(GL_TEXTURE_2D, 0);
}
void CPUNeighbours(bool *CAGrid, bool *NextCAGrid, int WorldH, int WorldW)
{
	int neighbours;

	for(int i =0; i < (WorldH) * (WorldW) ; i++)
	{
		int colup = i - WorldW;
		int coldwn = i + WorldW;
		int leftn = i - 1;
		int rightn = i + 1;
		if (colup < 0)colup = WorldW*(WorldH - 1) + i;
		if (coldwn > WorldH*WorldW)coldwn = i - WorldW*(WorldH-1);
		if (leftn < 0)leftn = i + WorldW;
		if (rightn > WorldW)rightn = i - WorldW;

		neighbours = CAGrid[rightn] +
			CAGrid[leftn] +
			CAGrid[colup - 1] +
			CAGrid[colup] +
			CAGrid[colup + 1] +
			CAGrid[coldwn - 1] +
			CAGrid[coldwn] +
			CAGrid[coldwn + 1];


		// END OF NEIGHBOUR ADDITION



		if (neighbours < 2)
		{
			NextCAGrid[i] = 0;
		}
		else if (neighbours > 3)
		{
			NextCAGrid[i] = 0;
		}
		else if (neighbours == 3)
		{
			NextCAGrid[i] = 1;
		}
	}
}

__global__ void NextGenKernel(bool *CAGrid, bool *NextCAGrid,int WorldH,int WorldW)
{
	//int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y*blockDim.y + threadIdx.y)* WorldH;
	int id = (blockIdx.y * gridDim.x + blockIdx.x);
	int neighbours = 0;
	unsigned int WorldS = WorldH*WorldW;
	
	unsigned int colup = ((blockIdx.y - 1) * gridDim.x + blockIdx.x);
	unsigned int coldwn = ((blockIdx.y + 1) * gridDim.x +blockIdx.x);
#ifndef ZeroBoundary
	if (blockIdx.y == 0) colup = ((gridDim.y - 1) * gridDim.x + blockIdx.x);
	if (blockIdx.y == gridDim.y-1) coldwn = blockIdx.x;
#endif
	//unsigned int colup = id - ((blockIdx.y - gridDim.x)*gridDim.x);
	//unsigned int coldwn = id + ((blockIdx.y + gridDim.x)*gridDim.x);
	//unsigned int colup = ((id - (id%WorldW)) + WorldS - WorldW) % WorldS;
	//unsigned int coldwn = ((id - (id%WorldW)) + WorldW) % WorldS;
	//unsigned int colup = id - ((blockIdx.y - 1)*blockDim.x);
	//unsigned int coldwn = id + ((blockIdx.y + 1)*blockDim.x);
	if (id < (WorldH) * (WorldW) && id>=0 )
	{
		// TODO WHY CODE 77 HERE - SOLVED ACCESS
		//neighbours = NeighboursEval_Global(CAGrid, id, WorldH, WorldW);
		// NEIGHBOUR ADDITION
		// TODO : colup and coldwn access violation? put zeros at boundaries
		// DOES THE CELL NEED TO BE ALIVE ? 
#ifdef ZeroBoundary




		if (colup <= 0)
		{ 
			if (id - 1 <= 0)
			{
				neighbours = CAGrid[id + 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];

			}
			else if (id + 1 > PanelW)
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];
			}
			else
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[id + 1] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];
			}
		}

		else if (coldwn > PanelH*PanelW)
		{
			if (id - 1 <= 0)
			{
				neighbours = CAGrid[id + 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
			else if (id + 1 > PanelW*PanelH)
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
			else
			{
				neighbours = CAGrid[id - 1] +
					CAGrid[id + 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1];
			}
		}

		//else if (id - 1 <= 0)
		//{
		//	neighbours = CAGrid[id + 1] +
		//		
		//		CAGrid[colup] +
		//		CAGrid[colup + 1] +
		//		
		//		CAGrid[coldwn] +
		//		CAGrid[coldwn + 1];
		//}
		//else if (id + 1 > PanelW)
		//{
		//	neighbours = CAGrid[id - 1] +
		//		CAGrid[colup - 1] +
		//		CAGrid[colup] +

		//		CAGrid[coldwn - 1] +
		//		CAGrid[coldwn];
		//		
		//}
		else
		{

			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];

		}
		// END OF NEIGHBOUR ADDITION
#else 
		if (id == 0)
		{
			if (colup == gridDim.x * gridDim.y)
			{
				neighbours = CAGrid[id + 1] +
					CAGrid[gridDim.x - 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup -gridDim.x] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];
			}
			else
			{
				neighbours = CAGrid[id + 1] +
					CAGrid[gridDim.x - 1] +
					CAGrid[colup - 1] +
					CAGrid[colup] +
					CAGrid[colup + 1] +
					CAGrid[coldwn - 1] +
					CAGrid[coldwn] +
					CAGrid[coldwn + 1];
			}
		}
		else if (id == gridDim.x*gridDim.y - 1)
		{

			neighbours = CAGrid[gridDim.x*gridDim.y -gridDim.x] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x*gridDim.y - gridDim.x)
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[gridDim.x-1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x - 1)
		{
			neighbours = CAGrid[id + 1]; +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup - gridDim.x + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else if (id == gridDim.x)
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[gridDim.x - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
		else
		{
			neighbours = CAGrid[id + 1] +
				CAGrid[id - 1] +
				CAGrid[colup - 1] +
				CAGrid[colup] +
				CAGrid[colup + 1] +
				CAGrid[coldwn - 1] +
				CAGrid[coldwn] +
				CAGrid[coldwn + 1];
		}
#endif // !ZeroBoundary

		if (CAGrid[id] == 1)
		{
			if (neighbours < 2)
			{
				NextCAGrid[id] = 0;
			}
			else if (neighbours > 3)
			{
				NextCAGrid[id] = 0;
			}
			else if (neighbours == 3 || neighbours == 2)
			{
				NextCAGrid[id] = 1;
			}
		}
		else
		{
			if (neighbours == 3) NextCAGrid[id] = 1;
		}
	}
}
__global__ void GLKernel(uchar4 *d_buf,bool *CAGrid,bool *NextCAGrid, int WorldH, int WorldW)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x);
	if (id < (WorldH ) * (WorldW ) && id >= 0)
	{

		if (CAGrid[id] == 1)
		{
			d_buf[id].w = 255;
			d_buf[id].x = 255;
			d_buf[id].y = 255;
			d_buf[id].z = 255;
		}
		if (CAGrid[id] == 0)
		{
			d_buf[id].w = 255;
			d_buf[id].x = 0;
			d_buf[id].y = 0;
			d_buf[id].z = 0;
		}
/*
			d_buf[id].w = 255;
			d_buf[id].x = 191;
			d_buf[id].y = 173;
			d_buf[id].z = 134;

		if (CAGrid[id] == 1 && NextCAGrid[id] == 1) //GETTING OLDER
		{
			d_buf[id].w = 255;
			d_buf[id].x -= 50;
			d_buf[id].y -= 50;
			d_buf[id].z -= 50;
		}
		if (CAGrid[id] == 1 && NextCAGrid[id] == 0) // NEW BORN
		{
			d_buf[id].w = 255;
			d_buf[id].x += 64;
			d_buf[id].y += 82;
			d_buf[id].z += 121;
		}
		if (CAGrid[id] ==0)	//DEAD
		{
			d_buf[id].w = 255;
			d_buf[id].x -= 191;
			d_buf[id].y -= 173;
			d_buf[id].z -= 134;
		}
		if (CAGrid[id] == 0 && NextCAGrid[id] == 1) // WAS ALIVE
		{
			d_buf[id].w = 255;
			d_buf[id].x += 50;
			d_buf[id].y += 50;
			d_buf[id].z += 50;
		}
		if (CAGrid[id] == 0 && NextCAGrid[id] == 0) // NO ONE
		{
			d_buf[id].w = 255;
			d_buf[id].x = 0;
			d_buf[id].y = 0;
			d_buf[id].z = 0;
		}
		*/
	}
}
void displayfunc()
{
	int WorldW = PanelW;
	int WorldH = PanelH;
	dim3 kernelwsize(WorldW, WorldH);
	
	dim3 kernelbsize(1);
	hipEvent_t start, stop; //CUDA timing var
	float ms;
	if (cont)
	{
#ifdef HEURISTICS
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
#endif
		NextGenKernel << <kernelwsize, kernelbsize >> > (d_CAGrid, d_next_CAGrid, WorldH, WorldW);
		//NextDumbKernel << <kernelwsize, kernelbsize  >> > (d_CAGrid, d_next_CAGrid);//,d_WorldH,d_WorldW);
		// Check for any errors launching the kernel

#ifdef HEURISTICS
		hipDeviceSynchronize();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ms, start, stop);
		printf(" Elapsed GPU Time: %f ms \n", ms);
#endif
		hipDeviceSynchronize();
		std::swap(d_CAGrid, d_next_CAGrid);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		

	}
	
	hipGraphicsMapResources(1, &cudaPboResource, 0);
	size_t num_bytes;
	hipError_t cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&GLout,
		&num_bytes, cudaPboResource);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Resource Mapping Error: %s\n", hipGetErrorString(cudaStatus));
		}
	//cudaGLSetGLDevice(0);
	GLKernel << < kernelwsize, kernelbsize >> > (GLout, d_CAGrid,d_next_CAGrid, WorldH, WorldW);
	hipGraphicsUnmapResources(1, &cudaPboResource, 0);
	drawTexture(WorldW, WorldH);
	glutSwapBuffers();
	//if (cont)glutPostRedisplay();
	//glutPostRedisplay();

}
int main(int argc,char** argv)
{
    

	const int WorldW = PanelW;
	const int WorldH = PanelH;
	const int WorldSize = WorldH * WorldW;
	bool *CAGrid = (bool *)calloc(WorldSize , sizeof(bool));
	bool *next_CAGrid = (bool *)calloc(WorldSize, sizeof(bool));
	const int reqGens = 1000;
	
	//IV
	int row = 0;
	for (int i =WorldH*2 /*(WorldH/2)*WorldW*/; i < WorldSize - WorldH*2; i++) //changed boundaries from 0 to worldsize to shown
	{
		//CAGrid[i] = rand() % 2;
		row = i / WorldW;
		if ((i > WorldW * row + WorldW / 5) && (i < WorldW*row + WorldW - WorldW / 5)) // our latest addition
		{
			if (row % 2 == 0)CAGrid[i] = 1;
			else CAGrid[i] = 0;
		}
#ifdef CPUGRAPHICS
		if (i % WorldH == 0)
		{
			printf("\n");
		}
		printf("%d", CAGrid[i]);
#endif
	}
#ifdef HEURISTICS
	clock_t start = clock(), diff;
	CPUNeighbours(CAGrid, next_CAGrid, WorldH, WorldW);
	diff = clock() - start;
	int msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("Time taken %d seconds %d milliseconds for CPU", msec / 1000, msec % 1000);
#endif
	hipError_t cudaStatus= CudaCAHelper(CAGrid, next_CAGrid, WorldSize,WorldH,WorldW,reqGens,&argc,argv);


	//glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_CONTINUE_EXECUTION);
	
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaCAHelper failed!");
        return 1;
    }
#ifdef CPUGRAPHICS
		for (int i = 0; i < WorldSize; i++)
		{
			if (i % WorldH == 0)
			{
				printf("\n");
			}
			printf("%d", CAGrid[i]);

		}
#endif
		printf("----------------------------");
		//system("CLS");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to do its magic 
hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size,unsigned int WorldH,unsigned int WorldW,unsigned int gen,int*argc,char**argv)
{
    hipError_t cudaStatus;
	dim3 kernelwsize(WorldW, WorldH);
	dim3 kernelbsize(1);

	hipDeviceProp_t myCUDA;
	if (hipGetDeviceProperties(&myCUDA, 0) == hipSuccess)
	{
		printf("Using device %d:\n", 0);
		printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
			myCUDA.name, (int)myCUDA.totalGlobalMem, (int)myCUDA.major,
			(int)myCUDA.minor, (int)myCUDA.clockRate);
		printf("Max Threads %d", myCUDA.maxThreadsPerBlock);
	}
	//int threadsPerBlock = myCUDA.maxThreadsPerBlock;
	//int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
	
	initGLUT(argc, argv, WorldW, WorldH);
	gluOrtho2D(0, WorldW, WorldH, 0); // VIewport
	glutKeyboardFunc(keyboard);
	glutSpecialFunc(handleSpecialKeypress);
	//glutPassiveMotionFunc(mouseMove);
	//glutMotionFunc(mouseDrag);
	glutDisplayFunc(displayfunc);	//Display function set
	OpenGLHelper(WorldW, WorldH);	//Texture and Buffer bind
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	/*bool *d_CAGrid; */cudaStatus = hipMalloc((void**)&d_CAGrid, sizeof(bool) *size); // ALLOCATE THE SAME MEMORY SIZE AS CPU FOR GPU 

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed with Grid!",cudaStatus);
        goto Error;
    }

	/*bool *d_next_CAGrid; */cudaStatus = hipMalloc((void**)&d_next_CAGrid, sizeof(bool) *size); // ALLOCATE THE SAME MEMORY SIZE AS CPU FOR GPU
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed with nextGrid!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
	//cudaStatus = hipMemcpy(d_WorldH, &WorldH,  sizeof(int), hipMemcpyHostToDevice);
    //cudaStatus = hipMemcpy(d_WorldW, &WorldW,  sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_CAGrid, CAGrid, size*sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	//REPLACED BY GLUT DISPLAY FUNC
	/*
	for (int i = 0; i < gen; i++)
	{
		NextGenKernel << <kernelwsize, kernelbsize >> > (d_CAGrid, d_next_CAGrid, WorldH, WorldW);
		//NextDumbKernel << <kernelwsize, kernelbsize  >> > (d_CAGrid, d_next_CAGrid);//,d_WorldH,d_WorldW);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NextGenKernel Start Error: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		std::swap(d_CAGrid, d_next_CAGrid);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching NextGenKernel!\n", cudaStatus);
			goto Error;
		}

		//Display Life Kernel Resource Mapping
		hipGraphicsMapResources(1, &cudaPboResource, 0);
		size_t num_bytes;
		cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&GLout,
			&num_bytes, cudaPboResource);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Resource Mapping Error: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		//cudaGLSetGLDevice(0);
		GLKernel << < kernelwsize, kernelbsize >> > (GLout, d_CAGrid, WorldH, WorldW);
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching NextGenKernel!\n", cudaStatus);
			goto Error;
		}
		hipGraphicsUnmapResources(1, &cudaPboResource, 0);
		drawTexture(WorldW, WorldH);
		glutSwapBuffers();
		

		
	}
	*/
	glutMainLoop();

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(CAGrid, d_CAGrid, size * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    
	}
	hipFree(d_CAGrid);
	hipFree(d_next_CAGrid);
	hipGraphicsUnregisterResource(cudaPboResource);
	glDeleteBuffers(1, &GLbufferID);
	glDeleteTextures(1, &GLtexture);
Error:
    hipFree(d_CAGrid);
    hipFree(d_next_CAGrid);
	hipGraphicsUnregisterResource(cudaPboResource);
	glDeleteBuffers(1, &GLbufferID);
	glDeleteTextures(1, &GLtexture);
    return cudaStatus;
}
