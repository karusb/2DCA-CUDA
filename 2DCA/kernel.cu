#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>  
#include <stdio.h>
#include <time.h>
#include <string>
#include <sstream>
#include "..\GL\glew.h"
#include "..\GL\freeglut.h"
#include "cuda_gl_interop.h"

// MANUALLY EDIT BELOW 
#define PanelW 500		// Texture Width
#define PanelH 500		// Texture Height
//#define ZeroBoundary		//if defined its only zero boundary at the 4 edges that defines the viewport
//

#include "Globals.hpp"
#include "GLCallbacks.hpp"
#include "HostFunctions.hpp"
#include "CudaHelpers.hpp"
#include "CellularAutomata.cuh"
//
/// CPUInsertGPU: Inserts a cell for the given location on the CPU and passes to GPU
/// Inputs: world width,height,window position of the chosen cell int2(x,y), pointer to CPU memory of the grid
/// IMPORTANT TODO : Location of the inserted cell is not accurate need to implement correct location scaling from window to texture
static void CPUInsertGPU(unsigned int WorldW, unsigned int WorldH, int2 i_loc, bool* CAGrid)
{
	float scalew = newpanelw / WorldW;
	float scaleh = newpanelh / WorldH;
	int myid = 0;
	if (scalew > 2.0 && scaleh > 2.0)
		myid = (i_loc.x / float(newpanelw / WorldW)) + (i_loc.y / float(newpanelh / WorldH)) * WorldW;
	else
		myid = i_loc.x + i_loc.y * newpanelw;
	if (myid <= WorldW * WorldH && myid > 0)CAGrid[myid] = !CAGrid[myid];
}

hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size, unsigned int WorldH, unsigned int WorldW, int*argc, char**argv);

///	OpenGLHelper: Initialises texture buffers i
/// Inputs: width and height of the texture
static void OpenGLHelper(unsigned int width,unsigned int height)
{
	glGenTextures(1, &GLtexture);
	glBindTexture(GL_TEXTURE_2D, GLtexture);
	// set basic parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// Create texture data (4-component unsigned byte)
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, d_texturedata);

	// Unbind the texture
	glBindTexture(GL_TEXTURE_2D, 0);

	glGenBuffers(1, &GLbufferID);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, GLbufferID);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, width * height * sizeof(uchar4), d_bufferdata, GL_STREAM_COPY);
	
	hipError_t result = hipGraphicsGLRegisterBuffer(&cudaPboResource, GLbufferID,
		cudaGraphicsMapFlagsWriteDiscard);
}
/// initGLUT: Initialises GLUT window
/// Inputs: Main function arguments argc,argv then window width,height
static bool initGLUT(int* argc, char** argv, unsigned int width,unsigned int height) {
	glutInit(argc, argv);  // Create GL context.
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(width, height);
	glutCreateWindow("2Dimensional Cellular Automata (Conways Game of Life)");

	glewInit();

	if (!glewIsSupported("GL_VERSION_2_0")) {
		printf( "ERROR: Support for necessary OpenGL extensions missing.\n");
		return false;
	}

	glutReportErrors();
	return true;
}

/// GLUT : displayfunc : Called everytime when screen update is called through GLUT, runs majority of the code as well as the CUDA kernels,
/// All of the global control variables are checked here
void displayfunc()
{
	int WorldW = PanelW;
	int WorldH = PanelH;
	dim3 kernelwsize(WorldW, WorldH);
	dim3 kernelbsize(1);

	glClear(GL_COLOR_BUFFER_BIT); //Clear color buf

	if(cont)
		NextGenKernel << <kernelwsize, kernelbsize >> > (d_CAGrid, d_next_CAGrid, WorldH, WorldW); // NextGeneration Kernel adds neighbours and sets nextCA grid

	hipDeviceSynchronize();
	if(cont)
		std::swap(d_CAGrid, d_next_CAGrid); //Swaps the values of both pointers

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	if (resetlife || fullresetlife)
	{
		if(resetlife)
			CPUGridInitRand(WorldW, WorldH, h_CAGrid, 5);
		if(fullresetlife)
			CPUGridInitFullRand(WorldW, WorldH, h_CAGrid);

		CudaReportOnError(
			hipMemset(d_CAGrid, 0, sizeof(d_CAGrid)), CACudaAction::Memset); // reset the current grid

		CudaReportOnError(
			hipMemcpy(d_CAGrid, h_CAGrid, WorldH * WorldW * sizeof(bool), hipMemcpyHostToDevice), CACudaAction::DeviceCopy);

		CudaReportOnError(
			hipMemset(d_next_CAGrid, 0, sizeof(d_next_CAGrid)), CACudaAction::Memset); // reset the next grid

		resetlife = false;
		fullresetlife = false;
	}
	if (givelife)
	{
		CudaReportOnError(
			hipMemcpy(h_CAGrid, d_CAGrid, WorldH * WorldW * sizeof(bool), hipMemcpyDeviceToHost), CACudaAction::HostCopy);

		CPUInsertGPU(WorldW,WorldH, loc2, h_CAGrid);

		CudaReportOnError(
			hipMemcpy(d_CAGrid, h_CAGrid, WorldH * WorldW * sizeof(bool), hipMemcpyHostToDevice), CACudaAction::DeviceCopy);

		givelife = false;
	}
	if(cont)
		evolution_number++;

	CudaReportOnError(
		hipGraphicsMapResources(1, &cudaPboResource, 0), CACudaAction::GraphicsResourceMap);

	size_t num_bytes;
	CudaReportOnError(
		hipGraphicsResourceGetMappedPointer((void**)&GLout, &num_bytes, cudaPboResource), CACudaAction::GraphicsResourceMap, "Texture mapping error: ");

	GLKernel << < kernelwsize, kernelbsize >> > (GLout, d_CAGrid,d_next_CAGrid, WorldH, WorldW,lifecontrol); //Fills GL texture with CA data
																							
	CudaReportOnError(
		hipGraphicsUnmapResources(1, &cudaPboResource, 0), CACudaAction::GraphicsResourceUnmap); //unmap resource memory
	 
	drawTexture(WorldW, WorldH); // call texture draw function

	glutSwapBuffers(); //swap back buffer with front buffer

	glFlush();
	if (evolutioncontrol)
		glutPostRedisplay(); //for consecutive frame update hence evolution if set
}

int main(int argc,char** argv)
{
	// BEGIN INFO
	printf("Starting GLUT main loop...\n");
	printf("Press [r] to reset the view to a randomized board \n")  ;
	printf("Press [f] to reset the view to a fully randomized board \n");
	printf("Press [ESC] to exit \n" ) ;
	printf("Press the [+] key to zoom in \n")  ;
	printf("Press the [-] key to zoom out \n")  ;
	printf("Press the [up arrow] to move up \n")  ;
	printf("Press the [down arrow] to move down \n")  ;
	printf("Press the [left arrow] to move left \n")  ;
	printf("Press the [right arrow] to move right \n")  ;
	printf("Press the [l] key to switch between colour and colourless \n")  ;
	printf("Press the [space] bar to stop evolution \n")  ;
	printf("Press the [e] key to evolve consecutively \n");
	printf("Press the [d] key to activate mouse functions \n");
	printf("MouseWheelUp = zoom+ - MouseWheelDown = zoom- \n");
	printf("MouseLeftClick and Drag to change viewing position \n");
	printf("MouseRightClick = Spawn or Kill a cell at the mouse location \n");

///// BEGIN GRID INIT
	CPUGridInitLine(WorldW, WorldH, h_CAGrid, 5);
//////
// 	
	// CUDA&GLUT Initialise Function
	hipError_t cudaStatus = CudaCAHelper(h_CAGrid, h_next_CAGrid, WorldSize, WorldH, WorldW, &argc, argv);
	
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaCAHelper failed!  CUDA Failure");
        return 1;
    }
	printf("----------------------------");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	CudaReportOnError(
		hipDeviceReset(), CACudaAction::Reset);

    return 0;
}

/// CudaCAHelper : Allocates Memory for CAGrid and NextCAGrid on GPU for given size
/// Inputs : Pointer to current CAGrid, pointer to NextCAGrid, world size (w*h), world height,width, required generations
hipError_t CudaCAHelper(bool *CAGrid, bool *NextCAGrid, unsigned int size,unsigned int WorldH,unsigned int WorldW, int*argc,char**argv)
{
	dim3 kernelwsize(WorldW, WorldH);
	dim3 kernelbsize(1);
	hipError_t status;
	hipDeviceProp_t myCUDA;
	status = CudaReportOnError(
		hipGetDeviceProperties(&myCUDA, 0), CACudaAction::GetProperty, "Is there Nvidia GPU Present?");

	if (status != hipSuccess)
		return status;

	printf("Using device %d:\n", 0);
	printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
		myCUDA.name, (int)myCUDA.totalGlobalMem, (int)myCUDA.major,
		(int)myCUDA.minor, (int)myCUDA.clockRate);
	printf("Max Threads %d", myCUDA.maxThreadsPerBlock);
    
	// Allocate GPU buffers for three vectors (two input, one output).
	status = CudaReportOnError(
		hipMalloc((void**)&d_CAGrid, sizeof(bool) * size), CACudaAction::Allocation, "First Grid");
	status = CudaReportOnError(
		hipMalloc((void**)&d_next_CAGrid, sizeof(bool) * size), CACudaAction::Allocation, "Second Grid", [&]() { hipFree(d_CAGrid); });

	if (status != hipSuccess)
		return status;

	initGLUT(argc, argv, WorldW, WorldH);
	gluOrtho2D(0, WorldW, WorldH, 0); // Viewport
	glutKeyboardFunc(keyboard);		//keyboard press func
	glutSpecialFunc(handleSpecialKeypress); //arrow keys
	glutMouseFunc(mouseCall);		// mouse clicks
	glutMotionFunc(mouseMove);		// mouse motion
	glutReshapeFunc(reshape);		// windows reshape function
	glutDisplayFunc(displayfunc);	//Display function
	OpenGLHelper(WorldW, WorldH);	//Texture and Buffer bind

	status = CudaReportOnError(
		hipSetDevice(0), CACudaAction::SetDevice, "Device Id 0", [&]() { hipFree(d_CAGrid); hipFree(d_next_CAGrid); });

    // Copy input vectors from host memory to GPU buffers.
	status = CudaReportOnError(
		hipMemcpy(d_CAGrid, CAGrid, size * sizeof(bool), hipMemcpyHostToDevice), CACudaAction::DeviceCopy);

	// BEGIN MAIN GLUT LOOP
	glutMainLoop();
	//
    // Copy output vector from GPU buffer to host memory.
	status = CudaReportOnError(
		hipMemcpy(CAGrid, d_CAGrid, size * sizeof(bool), hipMemcpyDeviceToHost), CACudaAction::HostCopy);

	// FREE ALLOCATED MEMORY ON LOOP EXIT
	hipFree(d_CAGrid);
	hipFree(d_next_CAGrid);
	hipGraphicsUnregisterResource(cudaPboResource);
	glDeleteBuffers(1, &GLbufferID);
	glDeleteTextures(1, &GLtexture);
    return status;
}
